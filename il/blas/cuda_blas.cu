#include "hip/hip_runtime.h"
// nvcc -std=c++11 cuda_blas.cu -o main -lcublas -lcurand

#include <cstdlib>
#include <cstdio>

#include <il/Timer.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

int main() {
  const int n = 16016;

  float* d_A;
  float* d_B;
  float* d_C;
  hipMalloc(&d_A, n * n * sizeof(float));
  hipMalloc(&d_B, n * n * sizeof(float));
  hipMalloc(&d_C, n * n * sizeof(float));

  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, static_cast<std::size_t>(clock()));
  hiprandGenerateUniform(prng, d_A, n * n);
  hiprandGenerateUniform(prng, d_B, n * n);

  il::Timer timer{};
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0f;
  float beta = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n,
              &beta, d_C, n);
  hipblasDestroy(handle);
  timer.stop();

  float* h_C = static_cast<float*>(malloc(n * n * sizeof(float)));
  hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);
  std::printf("Check: %8.4e\n", h_C[0]);
  free(h_C);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  long int m = n;
  std::printf("Time: %8.4e s\n", timer.time());
  std::printf("Gflops for CUDA: %8.4f Gflops\n", 1.0e-9 * 2 * m * m * m / timer.time());

  return 0;
}
